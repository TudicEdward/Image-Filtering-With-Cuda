#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Filtering_Functions.h"

__global__ void Negative_image(int h ,int w, unsigned char* Image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	

	if (x <= w && y <= h)
	{
		int tid = ((y * w) + x)*3;
		Image[tid] = 255 - Image[tid];
		Image[tid+1] = 255 - Image[tid+1];
		Image[tid+2] = 255 - Image[tid+2];
	}
	
}
__global__ void Grayscale_image(int h, int w, unsigned char* Image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x <= w && y <= h)
	{
		int tid = ((y * w) + x) * 3;
		Image[tid] = 0.299 * Image[tid] + 0.587 * Image[tid+1] + 0.114 * Image[tid+2];
		Image[tid + 1] = Image[tid];
		Image[tid + 2] = Image[tid];
	}
	
}
__global__ void Sepia_image(int h, int w, unsigned char* Image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x <= w && y <= h)
	{
		int tid = ((y * w) + x) * 3;
		int Red = 0.393 * Image[tid + 2] + 0.769 * Image[tid + 1] + 0.189 * Image[tid];
		int Green = 0.349 * Image[tid + 2] + 0.686 * Image[tid + 1] + 0.168 * Image[tid];
		int Blue = 0.272 * Image[tid + 2] + 0.534 * Image[tid + 1] + 0.131 * Image[tid];
		Image[tid] = Blue;
		Image[tid + 1] = Green;
		Image[tid + 2] = Red;
	}
	
}
__global__ void Red_image(int h, int w, unsigned char* Image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x <= w && y <= h)
	{
		int tid = ((y * w) + x) * 3;
		Image[tid] = 0;
		Image[tid + 1] = 0;

	}
	
}
__global__ void Green_image(int h, int w, unsigned char* Image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x <= w && y <= h)
	{
		int tid = ((y * w) + x) * 3;
		Image[tid] = 0;
		Image[tid + 2] = 0;
	}
	
}
__global__ void Blue_image(int h, int w, unsigned char* Image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x <= w && y <= h)
	{
		int tid = ((y * w) + x) * 3;
		Image[tid + 1] = 0;
		Image[tid + 2] = 0;
	}
	
}


void Image_Negative(unsigned char* Image, int Height, int Width) {
	unsigned char* Uploaded_Image = NULL;
	dim3 blocks((Width / 16) + 1, (Height / 16) + 1);
	dim3 threads(16, 16);

	hipMalloc((void**)&Uploaded_Image, Height * Width * 3);

	hipMemcpy(Uploaded_Image, Image, Height * Width * 3, hipMemcpyHostToDevice);
	
	Negative_image <<<blocks, threads >>> (Height, Width, Uploaded_Image);

	hipMemcpy(Image, Uploaded_Image, Height * Width * 3, hipMemcpyDeviceToHost);

	hipFree(Uploaded_Image);
}

void Image_Grayscale(unsigned char* Image, int Height, int Width) {
	unsigned char* Uploaded_Image = NULL;
	dim3 blocks((Width / 16)+1, (Height / 16)+1);
	dim3 threads(16, 16);

	hipMalloc((void**)&Uploaded_Image, Height * Width * 3);

	hipMemcpy(Uploaded_Image, Image, Height * Width * 3, hipMemcpyHostToDevice);
	
	Grayscale_image << <blocks, threads >> > (Height, Width, Uploaded_Image);

	hipMemcpy(Image, Uploaded_Image, Height * Width * 3, hipMemcpyDeviceToHost);

	hipFree(Uploaded_Image);
}

void Image_Sepia(unsigned char* Image, int Height, int Width) {
	unsigned char* Uploaded_Image = NULL;
	dim3 blocks((Width / 16) + 1, (Height / 16) + 1);
	dim3 threads(16, 16);

	hipMalloc((void**)&Uploaded_Image, Height * Width * 3);

	hipMemcpy(Uploaded_Image, Image, Height * Width * 3, hipMemcpyHostToDevice);

	Sepia_image << <blocks, threads >> > (Height, Width, Uploaded_Image);

	hipMemcpy(Image, Uploaded_Image, Height * Width * 3, hipMemcpyDeviceToHost);

	hipFree(Uploaded_Image);
}

void Image_Red(unsigned char* Image, int Height, int Width) {
	unsigned char* Uploaded_Image = NULL;
	dim3 blocks((Width / 16) + 1, (Height / 16) + 1);
	dim3 threads(16, 16);

	hipMalloc((void**)&Uploaded_Image, Height * Width * 3);

	hipMemcpy(Uploaded_Image, Image, Height * Width * 3, hipMemcpyHostToDevice);

	Red_image << <blocks, threads >> > (Height, Width, Uploaded_Image);

	hipMemcpy(Image, Uploaded_Image, Height * Width * 3, hipMemcpyDeviceToHost);

	hipFree(Uploaded_Image);
}

void Image_Green(unsigned char* Image, int Height, int Width) {
	unsigned char* Uploaded_Image = NULL;
	dim3 blocks((Width / 16) + 1, (Height / 16) + 1);
	dim3 threads(16, 16);

	hipMalloc((void**)&Uploaded_Image, Height * Width * 3);

	hipMemcpy(Uploaded_Image, Image, Height * Width * 3, hipMemcpyHostToDevice);

	Green_image << <blocks, threads >> > (Height, Width, Uploaded_Image);

	hipMemcpy(Image, Uploaded_Image, Height * Width * 3, hipMemcpyDeviceToHost);

	hipFree(Uploaded_Image);
}

void Image_Blue(unsigned char* Image, int Height, int Width) {
	unsigned char* Uploaded_Image = NULL;
	dim3 blocks((Width / 16) + 1, (Height / 16) + 1);
	dim3 threads(16, 16);

	hipMalloc((void**)&Uploaded_Image, Height * Width * 3);

	hipMemcpy(Uploaded_Image, Image, Height * Width * 3, hipMemcpyHostToDevice);

	Blue_image << <blocks, threads >> > (Height, Width, Uploaded_Image);

	hipMemcpy(Image, Uploaded_Image, Height * Width * 3, hipMemcpyDeviceToHost);

	hipFree(Uploaded_Image);
}